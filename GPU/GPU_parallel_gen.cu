
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define CUDA_SAFE_CALL( call ) {                                         \
 hipError_t err = call;                                                 \
 if( hipSuccess != err ) {                                              \
   fprintf(stderr,hipGetErrorString(err)); \
   exit(err);                                                            \
 } }

int leerMat(int, int, double *, const char *);
void imprimirMat(int, int, double *);
void convolucion(const double*, int, int, const double*, int, int, double*);

__global__ void compute_kernel( unsigned int mB, unsigned int nB,unsigned int mF,unsigned int nF, unsigned int nA, double *d_A, double *d_F, double *d_B ) {

  /* Obtain the global matrix index accessed by the thread executing this kernel */
  int i = blockIdx.x * blockDim.x + threadIdx.x; // Global row index
  int j = blockIdx.y * blockDim.y + threadIdx.y; // Global column index

  /* Perform the addition. Pay attention because probably not all the threads should perform the addition */ 
  if( i<mB && j<nB ) {
    // Reseteamos el valor acumulado
    double sum = 0.0;
    // Doble bucle para hacer el producto elemento a elemento
    // entre la submatriz de A y el filtro
    for (int k=0; k < mF; k++) {
        for (int s=0; s < nF; s++) {
            sum += d_A[(i+k)*nA + j + s] * d_F[k*nF + s];
        }
    }
    d_B[i*nB + j] = sum;
  } 
}


int main( int argc, char *argv[] ){

    if ( argc < 5 ){
        printf("\nUsage: %s mA nA mF nF [padding=0]\n", argv[0]);
        exit(-1);
    }
    int padding = 0;

    /* Padding */
    if ( (argc == 6) && (atoi(argv[5]) == 1) ) {
        padding = 1;
    }

    unsigned int mA, nA, mF, nF;
    mA = atoi(argv[1]);
    nA = atoi(argv[2]);
    mF = atoi(argv[3]);
    nF = atoi(argv[4]);

    unsigned int sizeA,sizeF;
    sizeA = mA * nA;
    sizeF = mF * nF;

    /* Reserva de memoria */
    double *A, *F;
    A = (double *) malloc(sizeof(double) * sizeA);
    F = (double *) malloc(sizeof(double) * sizeF);

    /* Generación de las matrices A y F */
    for (int i=0; i < mA; i++){
        for (int j=0; j < nA; j++){
            A[i*nA + j] = drand48();
        }
    }

    for (int i=0; i < mF; i++){
        for (int j=0; j < nF; j++){
            F[i*nF + j] = drand48();
        }
    }

    // printf("\n******* Matriz A ********\n");
    // imprimirMat(mA, nA, A);

    // printf("\n******* Matriz F ********\n");
    // imprimirMat(mF, nF, F);


    /* Reserva de memoria de B, en función de si el padding está activado */
    double *B;
    int mB, nB, sizeB;

    if ( padding ) {
        mB = mA;
        nB = nA;
        sizeB = sizeA;
    } else {
        mB = mA - mF + 1;
        nB = nA - nF + 1;
        sizeB = mB * nB;
    }

    B = (double *) malloc(sizeof(double) * sizeB);


    /* ****** Cálculo de convolución ****** */ 

    convolucion(A, mA, nA, F, mF, nF, B);

    /* ************************************ */

    // printf("\n******* Matriz B ********\n");
    // imprimirMat(mB, nB, B);
    FILE *temp=fopen("temp.txt","w");
    if(temp == NULL)
    {
      printf("Error!");   
      exit(1);             
    }
    for(int i=0; i<nB; i++){
        for(int j=0; j<mB; j++){
            double val = B[i*nB + j];
            fprintf(temp,"%f", val);
            fprintf(temp,"%s", " ");
        }
        fprintf(temp,"\n");
    }
    fclose(temp);


    // // Calculo del error acumulado
    // double err = 0.0;
    // for (int i=0; i < tamB; i++){
    //     for (int j=0; j < tamB; j++){
    //         err += abs(B[i*tamB + j] - Bsol[i*tamB + j]);
    //     }
    // }

    // printf("\nError acumulado = %lf\n", err);
    // printf("\nTiempo =  %lf segundos\n", elapsed);



    free(A);
    free(F);
    free(B);
    return 0;
}

/* Algoritmo de convolución */
void convolucion(const double *A, int mA, int nA, const double *F, int mF, int nF, double *B){
    /*
        Cada elemento de B de calcula como el producto elemento elemento
        de una submatriz de A y el Filtro, siendo la submatriz de A del
        tamaño que el filtro
    */

    // Tamaño de B, se puede generalizar
    int mB = mA - mF + 1;
    int nB = nA - nF + 1;

    unsigned int mem_size_A = mA * nA * sizeof(double);
    unsigned int mem_size_F = mF * nF * sizeof(double);
    unsigned int mem_size_B = mB * nB * sizeof(double);

    printf("Allocating memory\n");

    double *d_A, *d_F, *d_B;
    CUDA_SAFE_CALL( hipMalloc((void **) &d_A, mem_size_A ) );
    CUDA_SAFE_CALL( hipMalloc((void **) &d_F, mem_size_F ) );
    CUDA_SAFE_CALL( hipMalloc((void **) &d_B, mem_size_B ) );

    printf("Coping HOST to DEVICE\n");

    CUDA_SAFE_CALL( hipMemcpy( d_A, A, mem_size_A, hipMemcpyHostToDevice ) );
    CUDA_SAFE_CALL( hipMemcpy( d_F, F, mem_size_F, hipMemcpyHostToDevice ) );
    CUDA_SAFE_CALL( hipMemcpy( d_B, B, mem_size_B, hipMemcpyHostToDevice ) );

    unsigned int nthreadsX = 8;
    unsigned int nthreadsY = ceil(nthreadsX*mB/nB);

    unsigned int nblocksX = ceil(mB/nthreadsX) + 1;
    unsigned int nblocksY = ceil(nB/nthreadsY) + 1;

    printf("nthreadsX: %d\n", nthreadsX);
    printf("nthreadsY: %d\n", nthreadsY);

    printf("nblocksX: %d\n", nblocksX);
    printf("nblocksY: %d\n", nblocksY);

    dim3 dimGrid( nblocksX, nblocksY );
    dim3 dimBlock( nthreadsX, nthreadsY );

    printf("Executing kernel\n");

    hipEvent_t start, stop;
    float elapsedTime;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    compute_kernel<<< dimGrid, dimBlock >>>( mB, nB, mF, nF, nA, d_A, d_F, d_B );

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start,stop);
    printf("Time = %f ms\n",elapsedTime);

    printf("Coping DEVICE to HOST\n");

    CUDA_SAFE_CALL( hipMemcpy( B, d_B, mem_size_B, hipMemcpyDeviceToHost ) );

    // Deallocate device memory
    CUDA_SAFE_CALL( hipFree(d_A) );
    CUDA_SAFE_CALL( hipFree(d_F) );
    CUDA_SAFE_CALL( hipFree(d_B) );
}

/* La matriz se almacena por filas */
int leerMat(int m, int n, double *mat, const char *filename){

    FILE *file = fopen(filename, "r");

    if (file == NULL){
        return 1;
    }

    for (int i=0; i < m; i++){
        for (int j=0; j < n; j++){

            if (j == n-1) {
                fscanf(file, "%lf", &mat[i*n + j]);

            } else {
                fscanf(file, "%lf,", &mat[i*n + j]);
            }
            
        }
    }

    fclose(file);

    return 0;
}

/* Almacenada por filas */
void imprimirMat(int m, int n, double *mat){
    for (int i=0; i < m; i++) {
        for (int j=0; j < n; j++) {
            printf("%lf ", mat[i*n + j]);
        }
        printf("\n");
    }
}

